#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 
 
/*
 Compile:
  nvcc -arch=sm_52 -O3 -lcublas -lcurand -o LSTM LSTM.cu 
  
  To enable/disable different performance options add the flat -DPERFOPTSx
  Where x is a bitmask defining the options used (see below).
  
 Run:
  ./LSTM
  or
  ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch>
  
 Example (run on an NVIDIA M40):
   > ./LSTM
   Running with default settings
   seqLength 100, numLayers 4, hiddenSize 512, miniBatch 64
   i checksum (example 0) 5.113463E+04
   h checksum (example 0) 2.048000E+03
   c checksum (example 0) 2.058137E+05
   i checksum 3.272639E+06     c checksum 1.317278E+07     h checksum 1.310720E+05
   Runtime 27.807743ms
*/

#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>

// Performance is not significantly different, but false saves memory. 
// False does not work with unfused pointwise ops.
#define TRAINING (true)
// #define PEEPHOLES 

#define HFUNC tanhf
#define DEHFUNC de_tanhf
#define GFUNC tanhf
#define DEGFUNC de_tanhf

#ifndef PERFOPTS
   #define PERFOPTS (31)
#endif

#define GROUP_GEMM ((PERFOPTS & 1))
#define USE_STREAMS ((PERFOPTS & 2))
#define FUSE_PW ((PERFOPTS & 4))
#define PRE_TRANSPOSE ((PERFOPTS & 8))
#define RECUR_BATCH_SIZE (((PERFOPTS & 16) ? 2 : 1))
#define RECUR_BATCH_BP_SIZE (((PERFOPTS & 16) ? 4 : 1))

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}



// Device functions
__forceinline__ __device__ float sigmoidf(float in) {
   return 1.f / (1.f + expf(-in));  
}

__forceinline__ __device__ float de_sigmoidf(float out) {
   return out * (1-out);
} 

__forceinline__ __device__ float linearf(float in) {
  return in;
}

__forceinline__ __device__ float de_linearf(float out) {
  return 1.f;
}

__forceinline__ __device__ float de_tanhf(float out) {
   return 1.f - pow(out, 2);
}

__global__ void pw_de_tanh(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = 1 - pow(a[i], 2);
} 

__global__ void pw_de_sigmoid(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] * (1 - a[i]);
} 
// Pointwise functions
__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] += bias[i % nBias];
}

__global__ void pw_peepsAdd(float *y, float *peeps, float *x, int n, int nPeeps) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] += peeps[i % nPeeps] * x[i];
}

__global__ void pw_vecAdd(float *y, float *a,  float *b, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, float *a,  float *b, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] * b[i];
}

__global__ void pw_tanh(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = sigmoidf(a[i]);
}



// Unfused LSTM (calling many pointwise kernels).
int LSTM_elementwise_unfused( int hiddenSize, 
                               int miniBatch,
                               float * __restrict__ tmp_h, 
                               float * __restrict__ tmp_i, 
                               float * __restrict__ bias,
                               float * __restrict__ peeps,
                               // float * __restrict__ linearGates,
                               // float * __restrict__ h_data,
                               float * __restrict__ i_data,
                               float * __restrict__ c_in,
                               float * __restrict__ c_out,
                               bool training,
                               hipStream_t stream) {
  dim3 blockDim;
  dim3 gridDim;

  int numElements = hiddenSize * miniBatch;

  blockDim.x = 128;
  gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

               
  for (int i = 0; i < 4; i++) {
    if (tmp_h != NULL) {
       pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, tmp_i  + i * numElements, tmp_h  + i * numElements, numElements);
       cudaErrCheck(hipGetLastError());
    }

    pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, bias + i       * hiddenSize, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());
    
    if (i == 0) {
       pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, bias + i       * hiddenSize, numElements, hiddenSize);
       cudaErrCheck(hipGetLastError());
    }
    
    if (training) {
       printf("LSTM_elementWise_unfused does not support training\n"); 
       return 1;
    }
  }

  float *in_gate     = tmp_i + 0 * numElements;//i
  float *forget_gate = tmp_i + 1 * numElements;//f
  float *in_gate2    = tmp_i + 2 * numElements;//z
  float *out_gate    = tmp_i + 3 * numElements;//o   


  if (c_in != NULL) {
    //i_t += p_i + c_t-1          
    pw_peepsAdd <<< gridDim, blockDim, 0, stream >>> (in_gate, peeps, c_in, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());
    //f_t += p_f + c_t-1          
    pw_peepsAdd <<< gridDim, blockDim, 0, stream >>> (forget_gate, peeps + 1 * hiddenSize, c_in, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());

  }


  pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (in_gate, tmp_i + 0 * numElements, numElements);
  cudaErrCheck(hipGetLastError());

  pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (forget_gate, tmp_i + 1 * numElements, numElements);
  cudaErrCheck(hipGetLastError());

  //z'
  pw_tanh    <<< gridDim, blockDim, 0, stream >>> (in_gate2, tmp_i + 2 * numElements, numElements);
  cudaErrCheck(hipGetLastError());

  if (c_in == NULL) {
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
    cudaErrCheck(hipGetLastError());
  } else {
    //f_t * c_t-1    
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (forget_gate, forget_gate, c_in, numElements);
    cudaErrCheck(hipGetLastError());
    
    //i_t * z
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
    cudaErrCheck(hipGetLastError());
    
    //c_t = f_t * c_t-1 + i_t * c_t'
    pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, forget_gate, numElements);
    cudaErrCheck(hipGetLastError());
  }

  //o_t += p_o * c_t  
  pw_peepsAdd <<< gridDim, blockDim, 0, stream >>> (out_gate, peeps + 2 * hiddenSize, in_gate, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());

  pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (out_gate, tmp_i + 3 * numElements, numElements);
  cudaErrCheck(hipGetLastError());



  if (c_out != NULL) {
    cudaErrCheck(hipMemcpyAsync(c_out, in_gate, numElements * sizeof(float), hipMemcpyDeviceToDevice, stream));
  }
  

  //tanh(c_t)
  pw_tanh <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, numElements);
  cudaErrCheck(hipGetLastError());

  //y = o_t * tanh(c_t)
  // pw_vecMul <<< gridDim, blockDim, 0, stream >>> (h_data, out_gate, in_gate, numElements);
  // cudaErrCheck(hipGetLastError());


  pw_vecMul <<< gridDim, blockDim, 0, stream >>> (i_data, out_gate, in_gate, numElements);
  cudaErrCheck(hipGetLastError());

  return 0;
}

// Fused forward kernel
__global__ void elementWise_fp(int hiddenSize, int miniBatch,
                               float *tmp_h, //hidden_size * mini_batch * 4: W*xt
                               float *tmp_i, //hidden_size * mini_batch * 4: R*yt
                               float *bias, //hidden_size * 4: b*
                               float *peeps,//hidden_size * 3: p*
                               // float *linearGates,// hidden_size * mini_batch * 4
                               float *stateGates,
                               // float *h_out, //h_data
                               float *i_out,
                               float *c_in,
                               float *c_out,
                               float *label,
                               float *loss,
                               float *mask,
                               float *y_diff,
                               bool training) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numElements = miniBatch * hiddenSize;

  if (index >= numElements) return;

  int batch = index / hiddenSize;
  int gateIndex = (index % hiddenSize) + 4 * batch * hiddenSize;   

  float g[4];

  for (int i = 0; i < 4; i++) {
    g[i] = tmp_i[i * hiddenSize + gateIndex] + (tmp_h == NULL? 0:tmp_h[i * hiddenSize + gateIndex]);
    g[i] += bias[i * hiddenSize + index % hiddenSize];
  }  

  #ifdef PEEPHOLES 

    g[0] += peeps[index % hiddenSize] * c_in[index];//pi * c[t-1]
    g[1] += peeps[hiddenSize + index % hiddenSize] * c_in[index];//pf * c[t-1]

  #endif

  float in_gate     = sigmoidf(g[0]);//i
  float forget_gate = sigmoidf(g[1]);//f
  float in_gate2    = GFUNC(g[2]);//z

  float val = (forget_gate * c_in[index]) + (in_gate * in_gate2);//c[t] = z*i+c[t-1]*f
  c_out[index] = val;

  #ifdef PEEPHOLES
    g[3] += peeps[hiddenSize*2 + index % hiddenSize] * c_out[index];//po * c[t]
  #endif

  float out_gate = sigmoidf(g[3]);//o

  // #ifdef TRAINING
  if(training) {
    stateGates[gateIndex] = in_gate;
    stateGates[hiddenSize + gateIndex] = forget_gate;
    stateGates[2*hiddenSize + gateIndex] = in_gate2;
    stateGates[3*hiddenSize + gateIndex] = out_gate;
  }
  // #endif

  val = out_gate * HFUNC(val); //h                            

  // h_out[index] = val;
  i_out[index] = val;

  if (label != NULL) {
    if (mask[index % hiddenSize] == 1) {

      loss[index] = pow(val - label[index], 2);
      if (training)
        y_diff[index] = 2*(val - label[index]);
    } else {
      if (training)
        y_diff[index] = 0;
    }
  }
}

// Fused backward kernel
__global__ void elementWise_bp(int hiddenSize, int miniBatch,
                                float *y_diff,
                                float *stateGates_diff_in,// hidden_size *  mini_batch * 4
                                float *stateGates_diff_out,// hidden_size * mini_batch * 4
                                float *stateGates,
                                float *peeps,
                                float *peeps_diff,
                                float *c_in,
                                float *c_out,
                                float *c_diff,
                                bool peeps_update) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numElements = miniBatch * hiddenSize;

  if (index >= numElements) return;

  int batch = index / hiddenSize;
  int gateIndex = (index % hiddenSize) + 4 * batch * hiddenSize;   
  

  float in_gate = stateGates[gateIndex];
  float forget_gate = stateGates[hiddenSize + gateIndex];
  float in_gate2 = stateGates[2 * hiddenSize + gateIndex];
  float out_gate = stateGates[3 * hiddenSize + gateIndex];

  float out_diff = y_diff[index]*HFUNC(c_out[index])*de_sigmoidf(out_gate); //do


  float peep_diff = 0;
  if (stateGates_diff_in != NULL) peep_diff += c_diff[index];
  #ifdef PEEPHOLES
      peep_diff += peeps[2 * hiddenSize + index % hiddenSize] * out_diff;//po * do
    if (stateGates_diff_in != NULL) { 

      peep_diff += peeps[ index % hiddenSize] * stateGates_diff_in[gateIndex] + //pi * di[t+1]
      peeps[ hiddenSize + index % hiddenSize] * stateGates_diff_in[hiddenSize + gateIndex];//pf * df[t+1]
    }
  #endif


  float local_c_diff = y_diff[index]*out_gate*DEHFUNC(HFUNC(c_out[index])) + peep_diff;
  float forget_diff = local_c_diff * c_in[index] * de_sigmoidf(forget_gate);
  float in_diff = local_c_diff * in_gate2 * de_sigmoidf(in_gate);
  float in_diff2 = local_c_diff * in_gate * DEGFUNC(in_gate2);

  stateGates_diff_out[gateIndex] = in_diff;
  stateGates_diff_out[hiddenSize + gateIndex] = forget_diff;
  stateGates_diff_out[2 * hiddenSize + gateIndex] = in_diff2;
  stateGates_diff_out[3 * hiddenSize + gateIndex] = out_diff;       

  #ifdef PEEPHOLES
    int peepIndex = (index % hiddenSize) + 3 * batch * hiddenSize;
    if (peeps_update) {
      peeps_diff[peepIndex] = in_diff * c_in[index];//p_i
      peeps_diff[hiddenSize + peepIndex] = forget_diff * c_in[index]; //p_f
    }
    peeps_diff[2 * hiddenSize + peepIndex] = out_diff * c_out[index]; //p_o
  #endif

  c_diff[index] = local_c_diff * forget_gate;
  
}



struct LSTM_scheduler
{
  // float *h_data;//y
  float *i_data;//x

  float *c_data;//c

  float *T;
  float *T_f;

  float *bias;

  

  float *tmp_h;
  float *tmp_i;
  // float *linearGates;
  float *stateGates;

  //diff
  float *stateGates_diff; //di,df,dz,do
  float *y_diff;//dy
  // float *T_diff;//dW, dR
  
  float *diff_helper;

  #ifdef PEEPHOLES
    float *peeps_diff;
    float *peeps;
  #endif

  float *c_diff;//dc*ft

  float *label;
  float *loss;
  float *mask;



  hipStream_t *stream_i;
  hipStream_t *stream_h;

  hipEvent_t **events_i;
  hipEvent_t **events_h;

  hipblasHandle_t handle;

  int hiddenSize;
  int miniBatch;
  int seqLength;
  int numLayers;
  int numElements;
  int inputSize;
  int inputNumElements;
  int inputLayerSize;
  int weightSize; 

  hipblasOperation_t transa;
  hipblasOperation_t transb;

  hiprandGenerator_t rng;
    

  void init_helper(float * device_ptr, float data, int size) {
    if (size == 0)
      return;
    float* host_ptr;
    host_ptr = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; ++i) {
      host_ptr[i] = data;
    }
    cudaErrCheck(hipMemcpy(device_ptr, host_ptr, size * sizeof(float), hipMemcpyHostToDevice));
  }


  void set_input(float * input) {

    cudaErrCheck(hipMemcpy(i_data, input, inputLayerSize * sizeof(float), hipMemcpyHostToDevice));
  }

  void set_label(float * label_) {
    
    cudaErrCheck(hipMemcpy(label, label_, numElements * seqLength * sizeof(float), hipMemcpyHostToDevice));
  }

  void set_mask(float * mask_) {
    if (mask_ == NULL) {
      init_helper(mask, 1, 1);
      init_helper(mask + 1, 0, hiddenSize-1);   
      
    }
    cudaErrCheck(hipMemcpy(mask, mask_, numElements * seqLength * sizeof(float), hipMemcpyHostToDevice)); 
  }

  void set_weight(float * T_f_, float * bias_, float * peeps_) {
    if (T_f_ != NULL) {
      cudaErrCheck(hipMemcpy(T_f, T_f_, weightSize * sizeof(float), hipMemcpyHostToDevice)); 
    }
    else {
      curandErrCheck(hiprandGenerateUniform(rng, T_f, weightSize));
    }
    
    if (bias_ != NULL) {
      cudaErrCheck(hipMemcpy(bias, bias_, hiddenSize * 4 * numLayers * sizeof(float), hipMemcpyHostToDevice)); 
    }
    else {
      curandErrCheck(hiprandGenerateUniform(rng, bias, hiddenSize * 4 * numLayers));
    }

    #ifdef PEEPHOLES
    if (peeps_!= NULL) {
      cudaErrCheck(hipMemcpy(peeps, peeps_, hiddenSize * 3 * numLayers * sizeof(float), hipMemcpyHostToDevice)); 
    }
    else {
      curandErrCheck(hiprandGenerateUniform(rng, bias, hiddenSize * 4 * numLayers));
    }
    #endif 
  }


  LSTM_scheduler(int hiddenSize_, int miniBatch_, int seqLength_, int numLayers_, int inputSize_)
  {
    transa = (PRE_TRANSPOSE && (seqLength > 1)) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    transb = HIPBLAS_OP_N;

    hiddenSize = hiddenSize_;
    miniBatch = miniBatch_;
    seqLength = seqLength_; 
    numLayers = numLayers_; 
    inputSize = inputSize_;
    
    numElements = hiddenSize * miniBatch;
    inputNumElements = inputSize * miniBatch;
    inputLayerSize = inputNumElements * seqLength;

    weightSize = inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4 + (numLayers - 1) * hiddenSize * hiddenSize * 8;


    cublasErrCheck(hipblasCreate(&handle));
    curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));


    stream_i = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
    stream_h = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
    
    // If we don't want to use streams we can launch everything in to the NULL stream
    for (int i = 0; i < numLayers; i++) {
       if (USE_STREAMS) {
          cudaErrCheck(hipStreamCreate(&stream_i[i]));
          // Priority is empirical.
          cudaErrCheck(hipStreamCreateWithPriority(&stream_h[i], 0, -1));   
       }
       else {
          stream_i[i] = NULL;  
          stream_h[i] = NULL;  
       }
    }
    
    
    events_i = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
    events_h = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
    for (int i = 0; i < numLayers; i++) {
       events_i[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
       events_h[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
    }

  }


  

  void init(float* mask_ = NULL) {
    
    // cudaErrCheck(hipMalloc((void**)&h_data, (seqLength + 1) * (inputNumElements + (numLayers - 1) * numElements) * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&i_data, (seqLength * inputNumElements + seqLength * numLayers  * numElements) * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&c_data, (seqLength + 1) * (numLayers) * numElements * sizeof(float)));

    

    cudaErrCheck(hipMalloc((void**)&T, (inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8)  * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&T_f, (inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8) * sizeof(float)));
    
    
    cudaErrCheck(hipMalloc((void**)&bias, numLayers * hiddenSize * 4 * sizeof(float)));

    cudaErrCheck(hipMalloc((void**)&label, numElements * seqLength * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&loss, numElements * seqLength * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&mask, hiddenSize * sizeof(float)));

    set_mask(mask_);
    init_helper(loss, 0, numElements * seqLength);
    init_helper(label, 1, numElements * seqLength);

    #ifdef PEEPHOLES
      cudaErrCheck(hipMalloc((void**)&peeps, numLayers * hiddenSize * 3 * sizeof(float)));
      init_helper(peeps, 0.1, numLayers * hiddenSize * 3);
      // curandErrCheck(hiprandGenerateUniform(rng, peeps, numLayers * hiddenSize * 3));
    #endif

    // Workspace
    cudaErrCheck(hipMalloc((void**)&tmp_h, 4 * numLayers * numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&tmp_i, 4 * seqLength * numElements * sizeof(float)));
    
    // // Activations
    if (TRAINING) {
       // cudaErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&stateGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&stateGates_diff, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&y_diff, seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&c_diff, numLayers * numElements * seqLength * sizeof(float)));
      
      

      #ifdef PEEPHOLES 
        cudaErrCheck(hipMalloc((void**)&peeps_diff, numLayers * numElements * seqLength * 3 * sizeof(float)));
        init_helper(peeps_diff, 0, numLayers * numElements * seqLength * 3 );
      #endif
      cudaErrCheck(hipMalloc((void**)&diff_helper, miniBatch * seqLength * sizeof(float)));

      init_helper(diff_helper, 1.0, miniBatch * seqLength);
      init_helper(c_diff, 0, numLayers * numElements);
      init_helper(stateGates_diff, 0, 4 * seqLength * numLayers * numElements);
    }

    // Initialise with random values.
        // curandErrCheck(hiprandGenerateUniform(rng, h_data, (seqLength + 1) * (numLayers) * numElements));
    // curandErrCheck(hiprandGenerateUniform(rng, c_data, (seqLength + 1) * (numLayers) * numElements));
    // cudaErrCheck(hipMemset(c_data, 0,  (seqLength + 1) * (numLayers) * numElements  * sizeof(float)));
    init_helper(c_data, 0, (seqLength + 1) * (numLayers) * numElements);
    // curandErrCheck(hiprandGenerateUniform(rng, i_data, seqLength * (inputNumElements + numLayers  * numElements)));
    // cudaErrCheck(hipMemset(i_data, 0, seqLength * (inputNumElements + numLayers  * numElements)  * sizeof(float)));
    init_helper(i_data, 0.2, seqLength * (inputNumElements + numLayers  * numElements));
    // curandErrCheck(hiprandGenerateUniform(rng, T_f, inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8));
    // cudaErrCheck(hipMemset(T_f, 0.1, inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8  * sizeof(float)));
    init_helper(T_f, 0.1, weightSize );

    // curandErrCheck(hiprandGenerateUniform(rng, bias, numLayers * hiddenSize * 4));
    init_helper(bias, 0.1, numLayers * hiddenSize * 4);



    
      
    

    if (TRAINING) {
      init_helper(y_diff, 0, seqLength*numLayers*numElements);
      init_helper(y_diff+seqLength*(numLayers-1)*numElements, 1, seqLength * numElements);
       // curandErrCheck(hiprandGenerateUniform(rng, y_diff+seqLength*(numLayers-1)*numElements, seqLength * numElements));
    }

    // curandErrCheck(hiprandDestroyGenerator(rng));


    // Make sure everything is done before we start the timers
    cudaErrCheck(hipDeviceSynchronize());
    // prepare T
    // float alpha = 1.f;
    // float beta = 0.f; 

    // for (int layer = 0; layer < numLayers; layer++) {                     
    //    float *T_i_in = T + layer * hiddenSize * hiddenSize * 8;
    //    float *T_i_out = T_f + layer * hiddenSize * hiddenSize * 8;

    //    float *T_h_in = T + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;
    //    float *T_h_out = T_f + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;

    //    cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
    //    cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4 * hiddenSize, hiddenSize, &alpha, T_i_in, hiddenSize, &beta, NULL, 4 * hiddenSize, T_i_out, 4 * hiddenSize));
     
    //    cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
    //    cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4 * hiddenSize, hiddenSize, &alpha, T_h_in, hiddenSize, &beta, NULL, 4 * hiddenSize, T_h_out, 4 * hiddenSize));
    // }  
    
  }

  void clearStates(float * input=NULL, float * label=NULL) {
    if(TRAINING) {

      if(input) set_input(input);
      if(label) set_label(label);

    }
      // init_helper(y_diff, 0, seqLength*(numLayers-1)*numElements);
      // init_helper(y_diff+seqLength*(numLayers-1)*numElements, 1, seqLength * numElements);
      // init_helper(peeps_diff, 0, 3 * numElements * numLayers * seqLength);

       // curandErrCheck(hiprandGenerateUniform(rng, y_diff+seqLength*(numLayers-1)*numElements, seqLength * numElements));
    // init_helper(c_diff, 0, numLayers * numElements );
    // init_helper(loss, 0, numElements * seqLength);
    cudaErrCheck(hipDeviceSynchronize());
  }

  float Forward(float* sumLoss) {

    float alpha = 1.f;
    float beta = 0.f; 

    float elapsedTime;
    hipEvent_t start, stop;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));

    cudaErrCheck(hipEventRecord(start));


    int lStart = 0;
    int lEnd = 0;
    int rStart = 0;
    int rEnd = 0;
    
    int recurBatchSize = RECUR_BATCH_SIZE;
    
    while (true) {
      // Many layer "scheduling".
      if (lEnd == 0) {
        lStart = 0;
        lEnd = 1;
        rStart = 0;
      }
      else {
        // Move "up" and "left"
        lStart++;
        lEnd++;
        
        rStart -= recurBatchSize;
        
        // Over the top or off the left, reset to layer 0
        if (lEnd > numLayers || rStart < 0) {
            rStart += (lStart + 1) * recurBatchSize;

            lStart = 0;
            lEnd = 1;
        }
        
        // Off the right, step up
        while (rStart >= seqLength && lEnd <= numLayers) {
            lStart++;
            lEnd++;
           
            rStart -= recurBatchSize;
        }
        
        
        // Over the top or off the left, done!
        if (lEnd > numLayers || rStart < 0) {
            break;
        }
      }

      rEnd = rStart + recurBatchSize;
      // printf("lStart %d lEnd %d rStart %d rEnd %d\n", lStart, lEnd,
        // rStart, rEnd);
      if (rEnd > seqLength) rEnd = seqLength;

      for (int layer = lStart; layer < lEnd; layer++) {         
        cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
        
        //wait for xt to be calculated
        for (int i = rStart; i < rEnd; i++) {
            if (layer > 0) {
              cudaErrCheck(hipStreamWaitEvent(stream_i[layer], events_h[layer - 1][i], 0));
              cudaErrCheck(hipEventDestroy(events_h[layer - 1][i]));
            }
        }

        // Optimization 1
        if (GROUP_GEMM) {
          //[4N * N] x [N * 2m] = [4N * 2m] 
            cublasErrCheck(hipblasSgemm(handle,
                       transa, transb,
                       4 * hiddenSize, miniBatch * (rEnd - rStart), layer > 0 ? hiddenSize: inputSize,
                       &alpha,
                       layer > 0 ? &T_f[4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize]:T_f,
                       transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                       layer > 0 ? i_data + rStart * numElements + (layer - 1) * seqLength * numElements + seqLength * inputNumElements: i_data + rStart * inputNumElements,
                       layer > 0 ? hiddenSize:inputSize,
                       &beta,
                       tmp_i + 4 * rStart * numElements,
                       4 * hiddenSize));
        }
        else {
          for (int igemm =0; igemm < 4; igemm++) {
            cublasErrCheck(hipblasSgemm(handle,
                     transa, transb,
                     hiddenSize, miniBatch * (rEnd - rStart), layer > 0 ? hiddenSize:inputSize,
                     &alpha,
                     layer > 0 ? &T_f[4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize + igemm * hiddenSize]:&T_f[igemm * hiddenSize],
                     transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                     layer > 0 ? i_data + rStart * numElements + (layer - 1) * seqLength * numElements + seqLength * inputNumElements: i_data + rStart * inputNumElements,
                     layer > 0 ? hiddenSize:inputSize,
                     &beta,
                     tmp_i + 4 * rStart * numElements + igemm * hiddenSize,
                     4 * hiddenSize)); 
          }
        }
        
        for (int i = rStart; i < rEnd; i++) {
          cudaErrCheck(hipEventCreate(&events_i[layer][i], hipEventDisableTiming));
          cudaErrCheck(hipEventRecord(events_i[layer][i], stream_i[layer]));  
        }            
        
        for (int i = rStart; i < rEnd; i++) {
          cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
          // Optimization 1
          if (GROUP_GEMM) {
            //[4N * N] x [N * m] = [4N * m] 
            if (i > 0) {
              cublasErrCheck(hipblasSgemm(handle,
                          transa, transb,
                          4 * hiddenSize, miniBatch, hiddenSize,
                          &alpha,
                          layer > 0? &T_f[4 * hiddenSize * hiddenSize + 4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize]:&T_f[4 * inputSize * hiddenSize], 
                          transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                          // h_data + i * numElements + layer * (seqLength + 1) * numElements,
                          i_data + layer * seqLength * numElements + seqLength * inputNumElements + (i - 1) * numElements,
                          hiddenSize,
                          &beta,
                          tmp_h + 4 * layer * numElements, 
                          4 * hiddenSize));
            }
          }
          else {
            if (i > 0) {
              for (int igemm =0; igemm < 4; igemm++) {
                cublasErrCheck(hipblasSgemm(handle,
                           transa, transb,
                           hiddenSize, miniBatch, hiddenSize,
                           &alpha,
                           layer > 0? &T_f[4 * hiddenSize * hiddenSize + 4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize + igemm * hiddenSize]:&T_f[4 * inputSize * hiddenSize + igemm * hiddenSize], 
                           transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                           i_data + layer * seqLength * numElements + seqLength * inputNumElements + (i - 1) * numElements,
                           hiddenSize,
                           &beta,
                           tmp_h + 4 * layer * numElements + igemm * hiddenSize, 
                           4 * hiddenSize));
              }
            }
          }

          cudaErrCheck(hipStreamWaitEvent(stream_h[layer], events_i[layer][i], 0));
          cudaErrCheck(hipEventDestroy(events_i[layer][i]));

          // Optimization 3
          if (FUSE_PW) {
            dim3 blockDim;
            dim3 gridDim;
            
            blockDim.x = 256;
            gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;               
            
            elementWise_fp <<< gridDim, blockDim , 0, stream_h[layer] >>> 
                   (hiddenSize, miniBatch,
                    i > 0 ? tmp_h + 4 * layer * numElements:NULL, 
                    tmp_i + 4 * i * numElements, 
                    bias + 4 * layer * hiddenSize,
                    #ifdef PEEPHOLES 
                      peeps + 3 * layer * hiddenSize,
                    #else
                      NULL,
                    #endif
                    // TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                    TRAINING ? stateGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                    // h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    i_data + i * numElements + layer  * seqLength * numElements + seqLength * inputNumElements,
                    c_data + i * numElements + layer * (seqLength + 1) * numElements,
                    c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    layer == numLayers - 1 ? label + i * numElements : NULL,
                    layer == numLayers - 1 ? loss + i * numElements: NULL,
                    mask,
                    y_diff + i * numElements + layer * seqLength * numElements,
                    TRAINING);

            cudaErrCheck(hipGetLastError());
          }
          else {
            LSTM_elementwise_unfused(hiddenSize, miniBatch,
                    i > 0 ? tmp_h + 4 * layer * numElements:NULL, 
                    tmp_i + 4 * i * numElements, 
                    bias + 4 * layer * hiddenSize,
                    #ifdef PEEPHOLES  
                      peeps + 3 * layer * hiddenSize,
                    #else
                      NULL,
                    #endif
                    // TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                    // h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    i_data + i * numElements + layer * seqLength * numElements + seqLength * inputNumElements,
                    c_data + i * numElements + layer * (seqLength + 1) * numElements,
                    c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    TRAINING,
                    stream_h[layer]);
          }
          if (layer != numLayers - 1) {
            cudaErrCheck(hipEventCreate(&events_h[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_h[layer][i], stream_h[layer]));  
          }
        }
      }
    } 

    cublasErrCheck(hipblasSetStream(handle, stream_h[numLayers - 1]));

    cublasErrCheck(hipblasSasum(handle, numElements * seqLength, 
      loss, 1, sumLoss));


    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&elapsedTime, start, stop));

    cudaErrCheck(hipDeviceSynchronize());
    cudaErrCheck(hipEventDestroy(start));
    cudaErrCheck(hipEventDestroy(stop));

    return elapsedTime;
  }

  float Backward(float learningRate) {

    float elapsedTime;
    hipEvent_t start_bp, stop_bp;
    cudaErrCheck(hipEventCreate(&start_bp));
    cudaErrCheck(hipEventCreate(&stop_bp));

    cudaErrCheck(hipEventRecord(start_bp));


    int lStart = 0;
    int lEnd = 0;
    int rStart = 0;
    int rEnd = 0;

    int rev_lStart = 0;
    int rev_lEnd = 0;
    int rev_rStart = 0;
    int rev_rEnd = 0;

    int recurBatchSize = RECUR_BATCH_BP_SIZE;
    
    while (true) {
       // Many layer "scheduling".
      if (lEnd == 0) {
        lStart = 0;
        lEnd = 1;
        rStart = 0; 
      } else {
        // Move "up" and "left"
        lStart++;
        lEnd++;
        
        rStart -= recurBatchSize;
        
        // Over the top or off the left, reset to layer 0
        if (lEnd > numLayers || rStart < 0) {
           rStart += (lStart + 1) * recurBatchSize;

           lStart = 0;
           lEnd = 1;
        }
        
        // Off the right, step up
        while (rStart >= seqLength && lEnd <= numLayers) {
           lStart++;
           lEnd++;
           
           rStart -= recurBatchSize;
        }
         
        // Over the top or off the left, done!
        if (lEnd > numLayers || rStart < 0) {
           break;
        }
      }

      rEnd = rStart + recurBatchSize;
      if (rEnd > seqLength) rEnd = seqLength;

      rev_lStart = numLayers - lEnd;
      rev_lEnd = numLayers - lStart;
      rev_rStart = seqLength - rStart - 1;
      rev_rEnd = seqLength - rEnd - 1;
      // printf("rev_lStart %d rev_lEnd %d rev_rStart %d rev_rEnd %d\n", rev_lStart, rev_lEnd, rev_rStart, rev_rEnd);
      

      for (int layer = rev_lStart; layer < rev_lEnd; layer++) {                 
          
        for (int i = rev_rStart; i > rev_rEnd; i--) {
          // printf("level %d row %d\n", layer, i);
          cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
          //wait for the upper layer
          if (layer < numLayers-1) {
            cudaErrCheck(hipStreamWaitEvent(stream_h[layer], events_h[layer + 1][i], 0));
            cudaErrCheck(hipEventDestroy(events_h[layer + 1][i]));
          }
          //pointwise operations get diff
          // cudaErrCheck(hipDeviceSynchronize());
          dim3 blockDim;
          dim3 gridDim;

          blockDim.x = 256;
          gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;               

          elementWise_bp <<< gridDim, blockDim , 0, stream_h[layer] >>> 
              (hiddenSize, miniBatch,
              y_diff + i * numElements + layer * numElements * seqLength, 
              (i == seqLength - 1) ? NULL : stateGates_diff + 4 * ((i + 1) * numElements + layer * seqLength * numElements), 
              stateGates_diff + 4 * (i * numElements + layer * seqLength * numElements), 
              stateGates + 4 * (i * numElements + layer * seqLength  * numElements),
              #ifdef PEEPHOLES
                peeps + 3 * layer * hiddenSize,
                peeps_diff + 3 * (i * numElements + layer * seqLength * numElements),
              #else
                NULL,
                NULL,
              #endif
              c_data + i * numElements + layer * (seqLength + 1) * numElements,
              c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
              c_diff + layer * numElements,
              i != 0);

          // printWeight();

          cudaErrCheck(hipGetLastError());

          if (i == 1) {
            cudaErrCheck(hipEventCreate(&events_i[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_i[layer][i], stream_h[layer])); 
          }

          if (i > 0) {
            //RT * diff = dy
            float alpha = 1.f;
            float beta = 1.f;

            if (GROUP_GEMM) {
              cublasErrCheck(hipblasSgemm(handle,
                    HIPBLAS_OP_T, transb,
                    hiddenSize, miniBatch, 4 * hiddenSize,
                    &alpha,
                    layer > 0 ? &T_f[(layer - 1) * 8 * hiddenSize * hiddenSize + 8 * hiddenSize * hiddenSize + 4 * hiddenSize * inputSize]:&T_f[4 * hiddenSize * inputSize], 
                    4 * hiddenSize,
                    stateGates_diff + 4 * (i * numElements + layer * seqLength  * numElements),
                    4 * hiddenSize,
                    &beta,
                    y_diff + layer * numElements * seqLength + (i - 1) * numElements, 
                    hiddenSize));
            }
            else {
              for (int igemm = 0; igemm < 4; igemm++) {
                
                  cublasErrCheck(hipblasSgemm(handle,
                    HIPBLAS_OP_T, transb,
                    hiddenSize, miniBatch, hiddenSize,
                    &alpha,
                    layer > 0 ? &T_f[(layer - 1) * 8 * hiddenSize * hiddenSize + 8 * hiddenSize * hiddenSize + 4 * hiddenSize * inputSize + igemm * hiddenSize]:&T_f[4 * hiddenSize * inputSize + igemm * hiddenSize], 
                    4 * hiddenSize,
                    stateGates_diff + 4 * (i * numElements + layer * seqLength * numElements) + igemm * hiddenSize,
                    4 * hiddenSize,
                    &beta,
                    y_diff + layer * numElements * seqLength + (i - 1) * numElements, 
                    hiddenSize));
                }
              }
            }
          
        }
          // transa = (PRE_TRANSPOSE && (seqLength > 1)) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

          //W*diff = dx
        int row = rev_rEnd+1;

        if (layer > 0) {
          float alpha = 1.f;
          float beta = 0.f; 

          if (GROUP_GEMM) {
            cublasErrCheck(hipblasSgemm(handle,
                      HIPBLAS_OP_T, transb,
                      hiddenSize, miniBatch*(rev_rStart - rev_rEnd), 4 * hiddenSize,
                      &alpha,
                      &T_f[4 * hiddenSize * inputSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize], 
                      4 * hiddenSize,
                      stateGates_diff + 4 * (row * numElements + layer * seqLength  * numElements),
                      4 * hiddenSize,
                      &beta,
                      y_diff + (layer - 1) * numElements * seqLength + row * numElements, 
                      hiddenSize));
          }
          else {
            for (int igemm = 0; igemm < 4; igemm++) {
              cublasErrCheck(hipblasSgemm(handle,
                      HIPBLAS_OP_T, transb,
                      hiddenSize, miniBatch*(rev_rStart - rev_rEnd), hiddenSize,
                      &alpha,
                      &T_f[4 * hiddenSize * inputSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize + igemm * hiddenSize], 
                      4 * hiddenSize,
                      stateGates_diff + 4 * (row * numElements + layer * seqLength  * numElements) + igemm * hiddenSize,
                      4 * hiddenSize,
                      &beta,
                      y_diff + (layer - 1) * numElements * seqLength + row * numElements, 
                      hiddenSize));
              beta = 1.f;
            }
          }
        }

        if(layer != 0) {
          for (int i = rev_rStart; i > rev_rEnd; i--) {
            cudaErrCheck(hipEventCreate(&events_h[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_h[layer][i], stream_h[layer])); 
          }
        } 

        if (row == 0) {
                
            float lr = -learningRate;

            float beta = 1.f;
            //update W
            cublasErrCheck(hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,
                        4 * hiddenSize, layer > 0 ? hiddenSize : inputSize, miniBatch * seqLength,
                        &lr,
                        stateGates_diff + 4 * layer * seqLength * numElements, 
                        4 * hiddenSize,
                        layer > 0 ? i_data + (layer - 1) * seqLength * numElements + seqLength * inputNumElements : i_data,
                        layer > 0 ? hiddenSize : inputSize,
                        &beta,
                        layer > 0 ? &T_f[4 * hiddenSize * inputSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize]:&T_f[0], 
                        4 * hiddenSize));


            cudaErrCheck(hipStreamWaitEvent(stream_i[layer], events_i[layer][1], 0));
            cudaErrCheck(hipEventDestroy(events_i[layer][1]));

            cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
            //update R
            cublasErrCheck(hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,
                        4 * hiddenSize, hiddenSize, miniBatch * (seqLength - 1),
                        &lr,
                        stateGates_diff + 4 *  (layer * seqLength  * numElements + numElements), 
                        4 * hiddenSize,
                        i_data + layer * seqLength * numElements + seqLength * inputNumElements,
                        hiddenSize,
                        &beta,
                        layer > 0 ? &T_f[(layer - 1) * 8 * hiddenSize * hiddenSize + 8 * hiddenSize * hiddenSize + 4 * hiddenSize * inputSize]:&T_f[4 * hiddenSize * inputSize], 
                        4 * hiddenSize));

            cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));

            //update bias
            cublasErrCheck(hipblasSgemv(handle,
                        HIPBLAS_OP_N, 
                        4 * hiddenSize, miniBatch * seqLength, 
                        &lr,
                        stateGates_diff + 4 *  (layer * seqLength  * numElements), 
                        4 * hiddenSize,
                        diff_helper,
                        1,
                        &beta,
                        &bias[layer * hiddenSize * 4], 
                        1));

            #ifdef PEEPHOLES
              //update peeps
            cublasErrCheck(hipblasSgemv(handle,
                        HIPBLAS_OP_N, 
                        3 * hiddenSize, miniBatch * seqLength, 
                        &lr,
                        peeps_diff + 3 *  (layer  * numElements * seqLength), 
                        3 * hiddenSize,
                        diff_helper,
                        1,
                        &beta,
                        &peeps[layer * hiddenSize * 3], 
                        1));
            #endif
            
          
        }
      }
    }
    cudaErrCheck(hipEventRecord(stop_bp));
    cudaErrCheck(hipEventSynchronize(stop_bp));
    cudaErrCheck(hipEventElapsedTime(&elapsedTime, start_bp, stop_bp));
    
    cudaErrCheck(hipDeviceSynchronize());
    cudaErrCheck(hipEventDestroy(start_bp));
    cudaErrCheck(hipEventDestroy(stop_bp));
    return elapsedTime;
  }

  void printChecksum() {

    float* testOutputi;
    // float* testOutputh;
    float* testOutputc;
    
    
    int numElements = hiddenSize * miniBatch;
    testOutputi = (float*)malloc(numElements * seqLength * sizeof(float));
    // testOutputh = (float*)malloc(numElements * numLayers * sizeof(float));
    testOutputc = (float*)malloc(numElements * numLayers * sizeof(float));

    cudaErrCheck(hipMemcpy(testOutputi, i_data + seqLength * inputNumElements + (numLayers-1) * seqLength * numElements, seqLength * numElements * sizeof(float), hipMemcpyDeviceToHost));
    for (int layer = 0; layer < numLayers; layer++) {
       // cudaErrCheck(hipMemcpy(testOutputh + layer * numElements, h_data + seqLength * numElements + layer * (seqLength + 1) * numElements, numElements * sizeof(float), hipMemcpyDeviceToHost));
       cudaErrCheck(hipMemcpy(testOutputc + layer * numElements, c_data + seqLength * numElements + layer * (seqLength + 1) * numElements, numElements * sizeof(float), hipMemcpyDeviceToHost));
    }
    double checksumi = 0.;
    // double checksumh = 0.;
    double checksumc = 0.;
    
    for (int m = 0; m < miniBatch; m++) {
       for (int j = 0; j < seqLength; j++) {
          for (int i = 0; i < hiddenSize; i++) {
             checksumi += testOutputi[j * numElements + m * hiddenSize + i];
             printf("i: (%d,%d): %f\n", j, i, testOutputi[j * numElements + m * hiddenSize + i]);
          }
       }
       for (int j = 0; j < numLayers; j++) {
          for (int i = 0; i < hiddenSize; i++) {         
             // checksumh += testOutputh[j * numElements + m * hiddenSize + i];
             checksumc += testOutputc[j * numElements + m * hiddenSize + i];
          }
       }
       
       if (m == 0) printf("i checksum (example %d) %E\n", m, checksumi);
       // if (m == 0) printf("h checksum (example %d) %E\n", m, checksumh);
       if (m == 0) printf("c checksum (example %d) %E\n", m, checksumc);
    }
    
    printf("i checksum %f     ", checksumi);
    printf("c checksum %E     \n", checksumc);
    // printf("h checksum %E\n", checksumh);
    
    free(testOutputi);
    free(testOutputc);
    // free(testOutputh);
    cudaErrCheck(hipDeviceSynchronize());
  }

  void printWeight() {
    int t_size = (4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize);
    int bias_size = 4 * hiddenSize * numLayers;
    int stats_size = 4 * numElements * seqLength;
    int c_diff_size = numLayers * numElements;
    

    float* t_output, * bias_output, * states_output, 
    * y_output, 
    * c_diff_output;

    t_output = (float*)malloc( t_size * sizeof(float));
    bias_output = (float*)malloc(bias_size * sizeof(float));
    states_output = (float*)malloc(8 * sizeof(float));
    y_output = (float*)malloc(2 * sizeof(float));
    c_diff_output = (float*)malloc(c_diff_size * sizeof(float));

    cudaErrCheck(hipMemcpy(t_output, T_f, t_size * sizeof(float),hipMemcpyDeviceToHost));  
    cudaErrCheck(hipMemcpy(bias_output,bias,bias_size * sizeof(float),hipMemcpyDeviceToHost)); 

    cudaErrCheck(hipMemcpy(states_output,stateGates_diff,stats_size * sizeof(float),hipMemcpyDeviceToHost));  
    cudaErrCheck(hipMemcpy(y_output, y_diff, seqLength * sizeof(float), hipMemcpyDeviceToHost));  
    cudaErrCheck(hipMemcpy(c_diff_output, c_diff, c_diff_size * sizeof(float), hipMemcpyDeviceToHost));  

    printf("weights:\t");
    for (int i = 0; i < t_size;i++) {
      printf("%f\t",t_output[i]);
    }
    printf("\nbias:\t");
    for (int i = 0; i < bias_size; i++) {
      printf("%f\t",bias_output[i]);
    }
    printf("\n");

    #ifdef PEEPHOLES
      float * peeps_output;
      int peeps_size = 3 * numElements * numLayers * seqLength;
      peeps_output = (float*)malloc(peeps_size * sizeof(float));
      cudaErrCheck(hipMemcpy(peeps_output,peeps_diff, peeps_size * sizeof(float),hipMemcpyDeviceToHost)); 
      printf("peeps:\t"); 
      for (int i = 0; i < peeps_size; i++) {
        printf("%f\t",peeps_output[i]);
      }
      printf("\n");
    #endif
      
    
    printf("states:\t");
    for (int i = 0; i < stats_size; i++) {
      printf("%f\t",states_output[i]);
    }
    printf("\ny:\t");
    for (int i = 0; i < seqLength; i++) {
      printf("%f\t",y_output[i]);
    }
    printf("\nc_diff:\t");
    for (int i = 0; i < c_diff_size ; i++) {
      printf("%f\t",c_diff_output[i]);
    }
    printf("\n");

  }

  void freeMemory() {
      // cudaErrCheck(hipFree(h_data));
    cudaErrCheck(hipFree(i_data));  
    cudaErrCheck(hipFree(c_data));  

    if (T != T_f) cudaErrCheck(hipFree(T)); 
    cudaErrCheck(hipFree(T_f));

    cudaErrCheck(hipFree(bias));

    cudaErrCheck(hipFree(loss));
    cudaErrCheck(hipFree(label));
    cudaErrCheck(hipFree(mask));
    

    #ifdef PEEPHOLES
      cudaErrCheck(hipFree(peeps));
    #endif
    cudaErrCheck(hipFree(tmp_h));
    cudaErrCheck(hipFree(tmp_i));


    if (TRAINING) {
      // cudaErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipFree(stateGates));
      cudaErrCheck(hipFree(stateGates_diff));
      cudaErrCheck(hipFree(y_diff));
      cudaErrCheck(hipFree(c_diff));
      
      
      #ifdef PEEPHOLES
        cudaErrCheck(hipFree(peeps_diff));
      #endif
      cudaErrCheck(hipFree(diff_helper));

    }

    for (int i = 0; i < numLayers; i++) {
      if (stream_i[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_i[i]));
      if (stream_h[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_h[i]));
    }

    free(stream_i);
    free(stream_h);

    for (int i = 0; i < numLayers; i++) {
      free(events_i[i]);
      free(events_h[i]);
    }

    free(events_i);
    free(events_h);
  }
   
};




float LSTMTest(int hiddenSize, int miniBatch, int seqLength, int numLayers, int inputSize, bool checkF) {

  float loss; 
  float elapsedTime;  

  // hipEvent_t global_start, global_end;
  // , run_start, run_end;
  // cudaErrCheck(hipEventCreate(&global_start));
  // cudaErrCheck(hipEventCreate(&global_end));
  // cudaErrCheck(hipEventCreate(&run_start));
  // cudaErrCheck(hipEventCreate(&run_end));
  // cudaErrCheck(hipEventDestroy(run_start));
  // cudaErrCheck(hipEventDestroy(run_end));



  LSTM_scheduler scheduler(hiddenSize,miniBatch,seqLength,numLayers,inputSize);


  scheduler.init();
  printf("Initialize success\n");
  // cudaErrCheck(hipEventRecord(global_start));


  

  // cudaErrCheck(hipEventRecord(run_start));
  // cudaErrCheck(hipEventSynchronize(run_start));
  
  // scheduler.Forward(&loss);
  // printf("Forward loss is %f\n", loss);


  // if (checkF) {
  //   scheduler.printChecksum();
  // }


  for (int i = 0; i < 10; i++) {
    elapsedTime = scheduler.Forward(&loss);
    printf("Forward time is %f, loss is %f\n", elapsedTime, loss);

    if (TRAINING) {
      // scheduler.clearStates();
      elapsedTime = scheduler.Backward(0.2);
      printf("Backward time is %f\n", elapsedTime);
    }

    // scheduler.printWeight();
  }

  // scheduler.Forward(&loss);
  // printf("Forward loss is %f\n", loss);

  // cudaErrCheck(hipEventRecord(run_end));
  // We're done. Print some checksums
  // if (checkF) {
  //   scheduler.printChecksum();
  // }
  
  // cudaErrCheck(hipEventRecord(global_end));
  // cudaErrCheck(hipEventSynchronize(global_end));
  
  
  
  
  
  // cudaErrCheck(hipEventElapsedTime(&elapsedTime, run_start, run_end));
  // printf("Running time used %f ms, avg %f\n", elapsedTime, elapsedTime/10);


  // cudaErrCheck(hipEventElapsedTime(&elapsedTime, global_start, global_end));
  // printf("Total time used %f ms\n", elapsedTime);


  scheduler.freeMemory();

  // cudaErrCheck(hipEventDestroy(global_start));
  // cudaErrCheck(hipEventDestroy(global_end));
  // cudaErrCheck(hipEventElapsedTime(&elapsedTime, global_start, run_start));
  // printf("Initialize time used %f ms\n", elapsedTime);

  // cudaErrCheck(hipEventElapsedTime(&elapsedTime, run_end, global_end));
  // printf("Memory free time used %f ms\n", elapsedTime);
  // cudaErrCheck(hipDeviceSynchronize());



  
  // LSTM_scheduler scheduler(hiddenSize,miniBatch,seqLength,numLayers,inputSize);


  // scheduler.init();
  // printf("Initialize success\n");

  // hipEvent_t global_start, global_end;
  // cudaErrCheck(hipEventCreate(&global_start));
  // cudaErrCheck(hipEventCreate(&global_end));

  // cudaErrCheck(hipEventRecord(global_start));
  // scheduler.Forward(&loss);
  // printf("Forward loss is %f\n", loss);


  // // if (checkF) {
  // //   scheduler.printChecksum();
  // // }


  // for (int i = 0; i < 10; i++) {
  //   if (TRAINING) {
  //     // scheduler.clearStates();
  //     elapsedTime = scheduler.Backward(0.2);
  //     printf("Backward time is %f\n", elapsedTime);
  //   }

  //   // scheduler.printWeight();

  //   // // Timing starts here
    
  //   elapsedTime = scheduler.Forward(&loss);
  //   printf("Forward time is %f, loss is %f\n", elapsedTime, loss);
  // }
  // // We're done. Print some checksums
  // // if (checkF) {
  // //   scheduler.printChecksum();
  // // }
  // cudaErrCheck(hipEventRecord(global_end));
  // cudaErrCheck(hipEventSynchronize(global_end));
  // cudaErrCheck(hipEventElapsedTime(&elapsedTime, global_start, global_end));
  
  // cudaErrCheck(hipDeviceSynchronize());
  
  // printf("Total time used %f ms\n", elapsedTime);

  // scheduler.freeMemory();

  return 0;
}

  

//   return 0;
// }


int main(int argc, char* argv[]) {

  int seqLength;
  int numLayers;
  int hiddenSize;
  int miniBatch; 
  int inputSize;

  printf("\n");   


  if (argc == 6) {
    seqLength = atoi(argv[1]);
    numLayers =  atoi(argv[2]);
    hiddenSize =  atoi(argv[3]);
    miniBatch =  atoi(argv[4]);   
    inputSize = atoi(argv[5]);
  }
  else if (argc == 1) {
    printf("Running with default settings\n");
    inputSize = 32;
    seqLength = 20;
    numLayers = 4;
    hiddenSize = 32;
    miniBatch = 64;
  }
  else {
    printf("Usage: ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch> <inputSize>\n");
    return 1;      
  }

  printf("seqLength %d, numLayers %d, hiddenSize %d, miniBatch %d inputSize %d\n", seqLength, numLayers, hiddenSize, miniBatch, inputSize);  

  int numRuns = 1;

  float totalTime = 0.f;
  for (int run = 0; run < numRuns; run++) {
    totalTime += LSTMTest(hiddenSize, miniBatch, seqLength, numLayers, inputSize, true);
  }

  // printf("Runtime %fms\n", totalTime / numRuns);
  return time < 0;
}

