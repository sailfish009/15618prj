#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 
 
/*
 Compile:
  nvcc -arch=sm_52 -O3 -lcublas -lcurand -o LSTM LSTM.cu 
  
  To enable/disable different performance options add the flat -DPERFOPTSx
  Where x is a bitmask defining the options used (see below).
  
 Run:
  ./LSTM
  or
  ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch>
  
 Example (run on an NVIDIA M40):
   > ./LSTM
   Running with default settings
   seqLength 100, numLayers 4, hiddenSize 512, miniBatch 64
   i checksum (example 0) 5.113463E+04
   h checksum (example 0) 2.048000E+03
   c checksum (example 0) 2.058137E+05
   i checksum 3.272639E+06     c checksum 1.317278E+07     h checksum 1.310720E+05
   Runtime 27.807743ms
*/

#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>

// Performance is not significantly different, but false saves memory. 
// False does not work with unfused pointwise ops.
#define TRAINING (true)
#define PEEPHOLES (false)
#define HFUNC tanhf
#define DEHFUNC de_tanhf
#define GFUNC tanhf
#define DEGFUNC de_tanhf
#ifndef PERFOPTS
   #define PERFOPTS (29)
#endif

#define GROUP_GEMM ((PERFOPTS & 1))
#define USE_STREAMS ((PERFOPTS & 2))
#define FUSE_PW ((PERFOPTS & 4))
#define PRE_TRANSPOSE ((PERFOPTS & 8))
#define RECUR_BATCH_SIZE (((PERFOPTS & 16) ? 2 : 1))

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}



// Device functions
__forceinline__ __device__ float sigmoidf(float in) {
   return 1.f / (1.f + expf(-in));  
}

__forceinline__ __device__ float de_sigmoidf(float out) {
   return out * (1-out);
} 

__forceinline__ __device__ float linearf(float in) {
  return in;
}

__forceinline__ __device__ float de_linearf(float out) {
  return 1.f;
}

__forceinline__ __device__ float de_tanhf(float out) {
   return 1.f - pow(out, 2);
}

__global__ void pw_de_tanh(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = 1 - pow(a[i], 2);
} 

__global__ void pw_de_sigmoid(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] * (1 - a[i]);
} 
// Pointwise functions
__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] += bias[i % nBias];
}

__global__ void pw_peepsAdd(float *y, float *peeps, float *x, int n, int nPeeps) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] += peeps[i % nPeeps] * x[i];
}

__global__ void pw_vecAdd(float *y, float *a,  float *b, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, float *a,  float *b, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] * b[i];
}

__global__ void pw_tanh(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = sigmoidf(a[i]);
}



// Unfused LSTM (calling many pointwise kernels).
int LSTM_elementwise_unfused( int hiddenSize, 
                               int miniBatch,
                               float * __restrict__ tmp_h, 
                               float * __restrict__ tmp_i, 
                               float * __restrict__ bias,
                               float * __restrict__ peeps,
                               // float * __restrict__ linearGates,
                               // float * __restrict__ h_data,
                               float * __restrict__ i_data,
                               float * __restrict__ c_in,
                               float * __restrict__ c_out,
                               bool training,
                               hipStream_t stream) {
  dim3 blockDim;
  dim3 gridDim;

  int numElements = hiddenSize * miniBatch;

  blockDim.x = 128;
  gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

               
  for (int i = 0; i < 4; i++) {
    if (tmp_h != NULL) {
       pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, tmp_i  + i * numElements, tmp_h  + i * numElements, numElements);
       cudaErrCheck(hipGetLastError());
    }

    pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, bias + i       * hiddenSize, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());
    
    if (i == 0) {
       pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, bias + i       * hiddenSize, numElements, hiddenSize);
       cudaErrCheck(hipGetLastError());
    }
    
    if (training) {
       printf("LSTM_elementWise_unfused does not support training\n"); 
       return 1;
    }
  }

  float *in_gate     = tmp_i + 0 * numElements;//i
  float *forget_gate = tmp_i + 1 * numElements;//f
  float *in_gate2    = tmp_i + 2 * numElements;//z
  float *out_gate    = tmp_i + 3 * numElements;//o   


  if (c_in != NULL) {
    //i_t += p_i + c_t-1          
    pw_peepsAdd <<< gridDim, blockDim, 0, stream >>> (in_gate, peeps, c_in, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());
    //f_t += p_f + c_t-1          
    pw_peepsAdd <<< gridDim, blockDim, 0, stream >>> (forget_gate, peeps + 1 * hiddenSize, c_in, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());

  }


  pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (in_gate, tmp_i + 0 * numElements, numElements);
  cudaErrCheck(hipGetLastError());

  pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (forget_gate, tmp_i + 1 * numElements, numElements);
  cudaErrCheck(hipGetLastError());

  //z'
  pw_tanh    <<< gridDim, blockDim, 0, stream >>> (in_gate2, tmp_i + 2 * numElements, numElements);
  cudaErrCheck(hipGetLastError());

  if (c_in == NULL) {
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
    cudaErrCheck(hipGetLastError());
  } else {
    //f_t * c_t-1    
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (forget_gate, forget_gate, c_in, numElements);
    cudaErrCheck(hipGetLastError());
    
    //i_t * z
    pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
    cudaErrCheck(hipGetLastError());
    
    //c_t = f_t * c_t-1 + i_t * c_t'
    pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, forget_gate, numElements);
    cudaErrCheck(hipGetLastError());
  }

  //o_t += p_o * c_t  
  pw_peepsAdd <<< gridDim, blockDim, 0, stream >>> (out_gate, peeps + 2 * hiddenSize, in_gate, numElements, hiddenSize);
    cudaErrCheck(hipGetLastError());

  pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (out_gate, tmp_i + 3 * numElements, numElements);
  cudaErrCheck(hipGetLastError());



  if (c_out != NULL) {
    cudaErrCheck(hipMemcpyAsync(c_out, in_gate, numElements * sizeof(float), hipMemcpyDeviceToDevice, stream));
  }
  

  //tanh(c_t)
  pw_tanh <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, numElements);
  cudaErrCheck(hipGetLastError());

  //y = o_t * tanh(c_t)
  // pw_vecMul <<< gridDim, blockDim, 0, stream >>> (h_data, out_gate, in_gate, numElements);
  // cudaErrCheck(hipGetLastError());


  pw_vecMul <<< gridDim, blockDim, 0, stream >>> (i_data, out_gate, in_gate, numElements);
  cudaErrCheck(hipGetLastError());

  return 0;
}

// Fused forward kernel
__global__ void elementWise_fp(int hiddenSize, int miniBatch,
                               float *tmp_h, //hidden_size * mini_batch * 4: W*xt
                               float *tmp_i, //hidden_size * mini_batch * 4: R*yt
                               float *bias, //hidden_size * 4: b*
                               float *peeps,//hidden_size * 3: p*
                               // float *linearGates,// hidden_size * mini_batch * 4
                               float *stateGates,
                               // float *h_out, //h_data
                               float *i_out,
                               float *c_in,
                               float *c_out,
                               float *label,
                               float *loss,
                               float *mask,
                               float *y_diff,
                               bool training) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numElements = miniBatch * hiddenSize;

  if (index >= numElements) return;

  int batch = index / hiddenSize;
  int gateIndex = (index % hiddenSize) + 4 * batch * hiddenSize;   

  float g[4];

  for (int i = 0; i < 4; i++) {
    g[i] = tmp_i[i * hiddenSize + gateIndex] + (tmp_h == NULL? 0:tmp_h[i * hiddenSize + gateIndex]);
    g[i] += bias[i * hiddenSize + index % hiddenSize];
  }  

  if (PEEPHOLES) {
    g[0] += peeps[index % hiddenSize] * c_in[index];//pi * c[t-1]
    g[1] += peeps[hiddenSize + index % hiddenSize] * c_in[index];//pi * c[t-1]
  }

  float in_gate     = sigmoidf(g[0]);//i
  float forget_gate = sigmoidf(g[1]);//f
  float in_gate2    = GFUNC(g[2]);//z

  float val = (forget_gate * c_in[index]) + (in_gate * in_gate2);//c[t] = z*i+c[t-1]*f
  c_out[index] = val;

  if (PEEPHOLES) {
    g[3] += peeps[hiddenSize*2 + index % hiddenSize] * c_out[index];//po * c[t]
  }

  float out_gate    = sigmoidf(g[3]);//o

  if(training) {
    
    stateGates[gateIndex] = in_gate;
    stateGates[hiddenSize + gateIndex] = forget_gate;
    stateGates[2*hiddenSize + gateIndex] = in_gate2;
    stateGates[3*hiddenSize + gateIndex] = out_gate;
  }

  val = out_gate * HFUNC(val); //h                            

  // h_out[index] = val;
  i_out[index] = val;

  if (label != NULL) {
    if (mask[index % hiddenSize] == 1) {

      loss[index] += pow(val - label[index], 2);
      if (training)
        y_diff[index] = 2*(val - label[index]);
    } 
  }
}

// Fused backward kernel
__global__ void elementWise_bp(int hiddenSize, int miniBatch,
                                float *y_diff,
                                float *stateGates_diff_in,// hidden_size *  mini_batch * 4
                                float *stateGates_diff_out,// hidden_size * mini_batch * 4
                                float *stateGates,
                                float *peeps,
                                float *peeps_diff,
                                float *c_in,
                                float *c_out,
                                float *c_diff,
                                bool peeps_update) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int numElements = miniBatch * hiddenSize;

  if (index >= numElements) return;

  int batch = index / hiddenSize;
  int gateIndex = (index % hiddenSize) + 4 * batch * hiddenSize;   


  float in_gate = stateGates[gateIndex];
  float forget_gate = stateGates[hiddenSize + gateIndex];
  float in_gate2 = stateGates[2 * hiddenSize + gateIndex];
  float out_gate = stateGates[3 * hiddenSize + gateIndex];

  float out_diff = y_diff[index]*HFUNC(c_out[index])*de_sigmoidf(out_gate); //do


  float peep_diff = 0;
  if (PEEPHOLES) {
    if (stateGates_diff_in != NULL) { 
      peep_diff += peeps[index % hiddenSize] * out_diff + peeps[hiddenSize + index % hiddenSize] * stateGates_diff_in[gateIndex] + peeps[2 * hiddenSize + index % hiddenSize] * stateGates_diff_in[hiddenSize + gateIndex];
    }
  }

  float local_c_diff = y_diff[index]*out_gate*DEHFUNC(HFUNC(c_out[index])) + peep_diff + c_diff[index];
  float forget_diff = local_c_diff * c_in[index] * de_sigmoidf(forget_gate);
  float in_diff = local_c_diff * in_gate2 * de_sigmoidf(in_gate);
  float in_diff2 = local_c_diff * in_gate * DEGFUNC(in_gate2);

  stateGates_diff_out[gateIndex] = in_diff;
  stateGates_diff_out[hiddenSize + gateIndex] = forget_diff;
  stateGates_diff_out[2 * hiddenSize + gateIndex] = in_diff2;
  stateGates_diff_out[3 * hiddenSize + gateIndex] = out_diff;       

  if (PEEPHOLES && peeps_update) {
    peeps_diff[gateIndex] += in_diff * c_in[index];//p_i
    peeps_diff[hiddenSize + gateIndex] += forget_diff * c_in[index]; //p_f
    peeps_diff[2 * hiddenSize + gateIndex] += out_diff * c_out[index]; //p_o
  }
  c_diff[index] = local_c_diff * forget_diff;
}



struct LSTM_scheduler
{
  // float *h_data;//y
  float *i_data;//x

  float *c_data;//c

  float *T;
  float *T_f;

  float *bias;
  float *peeps;

  float *tmp_h;
  float *tmp_i;
  // float *linearGates;
  float *stateGates;

  //diff
  float *stateGates_diff; //di,df,dz,do
  float *y_diff;//dy
  float *T_diff;//dW, dR
  float *bias_diff;
  float *diff_helper;
  float *peeps_diff;
  float *c_diff;//dc*ft

  float *label;
  float *loss;
  float *mask;

  hipStream_t *stream_i;
  hipStream_t *stream_h;

  hipEvent_t **events_i;
  hipEvent_t **events_h;

  hipblasHandle_t handle;

  int hiddenSize;
  int miniBatch;
  int seqLength;
  int numLayers;
  int numElements;
  int inputSize;
  int inputNumElements;

  hipblasOperation_t transa;
  hipblasOperation_t transb;

  void init_helper(float * device_ptr, float data, int size) {
    if (size == 0)
      return;
    float* host_ptr;
    host_ptr = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; ++i) {
      host_ptr[i] = data;
    }
    cudaErrCheck(hipMemcpy(device_ptr, host_ptr, size * sizeof(float), hipMemcpyHostToDevice));
  }





  LSTM_scheduler(int hiddenSize_, int miniBatch_, int seqLength_, int numLayers_, int inputSize_)
  {
    transa = (PRE_TRANSPOSE && (seqLength > 1)) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    transb = HIPBLAS_OP_N;

    hiddenSize = hiddenSize_;
    miniBatch = miniBatch_;
    seqLength = seqLength_; 
    numLayers = numLayers_; 
    inputSize = inputSize_;
    
    numElements = hiddenSize * miniBatch;
    inputNumElements = inputSize * miniBatch;
    cublasErrCheck(hipblasCreate(&handle));
    stream_i = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
    stream_h = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
    
    // If we don't want to use streams we can launch everything in to the NULL stream
    for (int i = 0; i < numLayers; i++) {
       if (USE_STREAMS) {
          cudaErrCheck(hipStreamCreate(&stream_i[i]));
          // Priority is empirical.
          cudaErrCheck(hipStreamCreateWithPriority(&stream_h[i], 0, -1));   
       }
       else {
          stream_i[i] = NULL;  
          stream_h[i] = NULL;  
       }
    }
    
    
    events_i = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
    events_h = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
    for (int i = 0; i < numLayers; i++) {
       events_i[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
       events_h[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
    }
  }


  

  void init() {
    
    // cudaErrCheck(hipMalloc((void**)&h_data, (seqLength + 1) * (inputNumElements + (numLayers - 1) * numElements) * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&i_data, (seqLength * inputNumElements + seqLength * numLayers  * numElements) * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&c_data, (seqLength + 1) * (numLayers) * numElements * sizeof(float)));

    

    cudaErrCheck(hipMalloc((void**)&T, (inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8)  * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&T_f, (inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8) * sizeof(float)));
    
    
    cudaErrCheck(hipMalloc((void**)&bias, numLayers * hiddenSize * 4 * sizeof(float)));

    cudaErrCheck(hipMalloc((void**)&label, numElements * seqLength * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&loss, numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&mask, hiddenSize * sizeof(float)));

    init_helper(mask, 1, 1);
    init_helper(mask + 1, 0, hiddenSize-1);
    init_helper(label, 1, numElements * seqLength);
    init_helper(loss, 0, numElements);

    if (PEEPHOLES) {
      cudaErrCheck(hipMalloc((void**)&peeps, numLayers * hiddenSize * 3 * sizeof(float)));
    }

    // Workspace
    cudaErrCheck(hipMalloc((void**)&tmp_h, 4 * numLayers * numElements * sizeof(float)));
    cudaErrCheck(hipMalloc((void**)&tmp_i, 4 * seqLength * numElements * sizeof(float)));
    
    // // Activations
    if (TRAINING) {
       // cudaErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&stateGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&stateGates_diff, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&y_diff, seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&c_diff, numLayers * numElements * seqLength * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&T_diff, numLayers * hiddenSize * hiddenSize * 8 * sizeof(float)));
      cudaErrCheck(hipMalloc((void**)&bias_diff, numLayers * hiddenSize * 4 * sizeof(float)));

      if (PEEPHOLES) {
        cudaErrCheck(hipMalloc((void**)&peeps_diff, numLayers * numElements * 3 * sizeof(float)));
      }
      cudaErrCheck(hipMalloc((void**)&diff_helper, miniBatch * seqLength * sizeof(float)));

      init_helper(diff_helper, 1.0, miniBatch * seqLength);
      init_helper(c_diff, 0, numLayers * numElements);
      init_helper(stateGates_diff, 0, 4 * seqLength * numLayers * numElements);
    }

    // Initialise with random values.
    // hiprandGenerator_t rng;
    // curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    // curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
    // curandErrCheck(hiprandGenerateUniform(rng, h_data, (seqLength + 1) * (numLayers) * numElements));
    // curandErrCheck(hiprandGenerateUniform(rng, c_data, (seqLength + 1) * (numLayers) * numElements));
    // cudaErrCheck(hipMemset(c_data, 0,  (seqLength + 1) * (numLayers) * numElements  * sizeof(float)));
    init_helper(c_data, 0, (seqLength + 1) * (numLayers) * numElements);
    // curandErrCheck(hiprandGenerateUniform(rng, i_data, seqLength * (inputNumElements + numLayers  * numElements)));
    // cudaErrCheck(hipMemset(i_data, 0, seqLength * (inputNumElements + numLayers  * numElements)  * sizeof(float)));
    init_helper(i_data, 0.2, seqLength * (inputNumElements + numLayers  * numElements));
    // curandErrCheck(hiprandGenerateUniform(rng, T_f, inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8));
    // cudaErrCheck(hipMemset(T_f, 0.1, inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8  * sizeof(float)));
    init_helper(T_f, 0.1, inputSize * hiddenSize * 4 + hiddenSize * hiddenSize * 4+ (numLayers - 1) * hiddenSize * hiddenSize * 8 );

    // curandErrCheck(hiprandGenerateUniform(rng, bias, numLayers * hiddenSize * 4));
    init_helper(bias, 0.1, numLayers * hiddenSize * 4);



    if (PEEPHOLES) {
      // curandErrCheck(hiprandGenerateUniform(rng, peeps, numLayers * hiddenSize * 3));
      init_helper(peeps, 0.1, numLayers * hiddenSize * 3);
    }

    if (TRAINING) {
      init_helper(y_diff, 0, seqLength*(numLayers-1)*numElements);
      init_helper(y_diff+seqLength*(numLayers-1)*numElements, 1, seqLength * numElements);
       // curandErrCheck(hiprandGenerateUniform(rng, y_diff+seqLength*(numLayers-1)*numElements, seqLength * numElements));
    }

    // curandErrCheck(hiprandDestroyGenerator(rng));


    // Make sure everything is done before we start the timers
    cudaErrCheck(hipDeviceSynchronize());
    // prepare T
    // float alpha = 1.f;
    // float beta = 0.f; 

    // for (int layer = 0; layer < numLayers; layer++) {                     
    //    float *T_i_in = T + layer * hiddenSize * hiddenSize * 8;
    //    float *T_i_out = T_f + layer * hiddenSize * hiddenSize * 8;

    //    float *T_h_in = T + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;
    //    float *T_h_out = T_f + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;

    //    cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
    //    cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4 * hiddenSize, hiddenSize, &alpha, T_i_in, hiddenSize, &beta, NULL, 4 * hiddenSize, T_i_out, 4 * hiddenSize));
     
    //    cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
    //    cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4 * hiddenSize, hiddenSize, &alpha, T_h_in, hiddenSize, &beta, NULL, 4 * hiddenSize, T_h_out, 4 * hiddenSize));
    // }  
    
  }

  void clearStates() {
    if (TRAINING) {
      init_helper(y_diff, 0, seqLength*(numLayers-1)*numElements);
      init_helper(y_diff+seqLength*(numLayers-1)*numElements, 1, seqLength * numElements);
      cudaErrCheck(hipDeviceSynchronize());
       // curandErrCheck(hiprandGenerateUniform(rng, y_diff+seqLength*(numLayers-1)*numElements, seqLength * numElements));
    }
  }

  float Forward(float* sumLoss) {

    float alpha = 1.f;
    float beta = 0.f; 

    float elapsedTime;
    hipEvent_t start, stop;
    cudaErrCheck(hipEventCreate(&start));
    cudaErrCheck(hipEventCreate(&stop));

    cudaErrCheck(hipEventRecord(start));


    int lStart = 0;
    int lEnd = 0;
    int rStart = 0;
    int rEnd = 0;
    
    int recurBatchSize = RECUR_BATCH_SIZE;
    
    while (true) {
      // Many layer "scheduling".
      if (lEnd == 0) {
        lStart = 0;
        lEnd = 1;
        rStart = 0;
      }
      else {
        // Move "up" and "left"
        lStart++;
        lEnd++;
        
        rStart -= recurBatchSize;
        
        // Over the top or off the left, reset to layer 0
        if (lEnd > numLayers || rStart < 0) {
            rStart += (lStart + 1) * recurBatchSize;

            lStart = 0;
            lEnd = 1;
        }
        
        // Off the right, step up
        while (rStart >= seqLength && lEnd <= numLayers) {
            lStart++;
            lEnd++;
           
            rStart -= recurBatchSize;
        }
        
        
        // Over the top or off the left, done!
        if (lEnd > numLayers || rStart < 0) {
            break;
        }
      }

      rEnd = rStart + recurBatchSize;
      // printf("lStart %d lEnd %d rStart %d rEnd %d\n", lStart, lEnd,
        // rStart, rEnd);
      if (rEnd > seqLength) rEnd = seqLength;

      for (int layer = lStart; layer < lEnd; layer++) {         
        cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
        
        //wait for xt to be calculated
        for (int i = rStart; i < rEnd; i++) {
            if (layer > 0) {
              cudaErrCheck(hipStreamWaitEvent(stream_i[layer], events_h[layer - 1][i], 0));
              cudaErrCheck(hipEventDestroy(events_h[layer - 1][i]));
            }
        }

        // Optimization 1
        if (GROUP_GEMM) {
          //[4N * N] x [N * 2m] = [4N * 2m] 
            cublasErrCheck(hipblasSgemm(handle,
                       transa, transb,
                       4 * hiddenSize, miniBatch * (rEnd - rStart), layer > 0 ? hiddenSize: inputSize,
                       &alpha,
                       layer > 0 ? &T_f[4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize]:T_f,
                       transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                       layer > 0 ? i_data + rStart * numElements + (layer - 1) * seqLength * numElements + seqLength * inputNumElements: i_data + rStart * inputNumElements,
                       layer > 0 ? hiddenSize:inputSize,
                       &beta,
                       tmp_i + 4 * rStart * numElements,
                       4 * hiddenSize));
        }
        else {
          for (int igemm =0; igemm < 4; igemm++) {
            cublasErrCheck(hipblasSgemm(handle,
                     transa, transb,
                     hiddenSize, miniBatch * (rEnd - rStart), layer > 0 ? hiddenSize:inputSize,
                     &alpha,
                     layer > 0 ? &T_f[4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize+ igemm * hiddenSize]:&T_f[igemm * hiddenSize],
                     hiddenSize,
                     layer > 0 ? i_data + rStart * numElements + (layer - 1) * seqLength * numElements + seqLength * inputNumElements: i_data + rStart * inputNumElements,
                     layer > 0 ? hiddenSize:inputSize,
                     &beta,
                     tmp_i + 4 * rStart * numElements + igemm * hiddenSize,
                     4 * hiddenSize)); 
          }
        }
        
        for (int i = rStart; i < rEnd; i++) {
          cudaErrCheck(hipEventCreate(&events_i[layer][i], hipEventDisableTiming));
          cudaErrCheck(hipEventRecord(events_i[layer][i], stream_i[layer]));  
        }            
        
        for (int i = rStart; i < rEnd; i++) {
          cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
          // Optimization 1
          if (GROUP_GEMM) {
            //[4N * N] x [N * m] = [4N * m] 
            if (i > 0) {
              cublasErrCheck(hipblasSgemm(handle,
                          transa, transb,
                          4 * hiddenSize, miniBatch, hiddenSize,
                          &alpha,
                          layer > 0? &T_f[4 * hiddenSize * hiddenSize + 4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize]:&T_f[4 * inputSize * hiddenSize], 
                          transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                          // h_data + i * numElements + layer * (seqLength + 1) * numElements,
                          i_data + layer * seqLength * numElements + seqLength * inputNumElements + (i - 1) * numElements,
                          hiddenSize,
                          &beta,
                          tmp_h + 4 * layer * numElements, 
                          4 * hiddenSize));
            }
          }
          else {
            if (i > 0) {
              for (int igemm =0; igemm < 4; igemm++) {
                cublasErrCheck(hipblasSgemm(handle,
                           transa, transb,
                           hiddenSize, miniBatch, hiddenSize,
                           &alpha,
                           layer > 0? &T_f[4 * hiddenSize * hiddenSize + 4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize + igemm * hiddenSize]:&T_f[4 * inputSize * hiddenSize + igemm * hiddenSize], 
                           transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                           i_data +  seqLength * inputNumElements + i * numElements + layer * seqLength * numElements,
                           hiddenSize,
                           &beta,
                           tmp_h + 4 * layer * numElements + igemm * hiddenSize, 
                           4 * hiddenSize));
              }
            }
          }

          cudaErrCheck(hipStreamWaitEvent(stream_h[layer], events_i[layer][i], 0));
          cudaErrCheck(hipEventDestroy(events_i[layer][i]));

          // Optimization 3
          if (FUSE_PW) {
            dim3 blockDim;
            dim3 gridDim;
            
            blockDim.x = 256;
            gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;               
            
            elementWise_fp <<< gridDim, blockDim , 0, stream_h[layer] >>> 
                   (hiddenSize, miniBatch,
                    i > 0 ? tmp_h + 4 * layer * numElements:NULL, 
                    tmp_i + 4 * i * numElements, 
                    bias + 4 * layer * hiddenSize,
                    PEEPHOLES ? peeps + 3 * layer * hiddenSize : NULL,
                    // TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                    TRAINING ? stateGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                    // h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    i_data + i * numElements + layer  * seqLength * numElements + seqLength * inputNumElements,
                    c_data + i * numElements + layer * (seqLength + 1) * numElements,
                    c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    layer == numLayers - 1 ? label + i * numElements : NULL,
                    loss,
                    mask,
                    y_diff + i * numElements + layer * seqLength * numElements,
                    TRAINING);

            cudaErrCheck(hipGetLastError());
          }
          else {
            LSTM_elementwise_unfused(hiddenSize, miniBatch,
                    i > 0 ? tmp_h + 4 * layer * numElements:NULL, 
                    tmp_i + 4 * i * numElements, 
                    bias + 4 * layer * hiddenSize,
                    PEEPHOLES ? peeps + 3 * layer * hiddenSize : NULL,
                    // TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                    // h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    i_data + i * numElements + layer * seqLength * numElements + seqLength * inputNumElements,
                    c_data + i * numElements + layer * (seqLength + 1) * numElements,
                    c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                    TRAINING,
                    stream_h[layer]);
          }
          if (layer != numLayers - 1) {
            cudaErrCheck(hipEventCreate(&events_h[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_h[layer][i], stream_h[layer]));  
          }
        }
      }
    } 

    cublasErrCheck(hipblasSetStream(handle, stream_h[numLayers - 1]));

    cublasErrCheck(hipblasSasum(handle, numElements, 
      loss, 1, sumLoss));


    cudaErrCheck(hipEventRecord(stop));
    cudaErrCheck(hipEventSynchronize(stop));
    cudaErrCheck(hipEventElapsedTime(&elapsedTime, start, stop));

    cudaErrCheck(hipDeviceSynchronize());
    return elapsedTime;
  }

  float Backward(float learningRate) {

    float elapsedTime;
    hipEvent_t start_bp, stop_bp;
    cudaErrCheck(hipEventCreate(&start_bp));
    cudaErrCheck(hipEventCreate(&stop_bp));

    cudaErrCheck(hipEventRecord(start_bp));


    int lStart = 0;
    int lEnd = 0;
    int rStart = 0;
    int rEnd = 0;

    int rev_lStart = 0;
    int rev_lEnd = 0;
    int rev_rStart = 0;
    int rev_rEnd = 0;

    int recurBatchSize = 1;
    
    
    while (true) {
       // Many layer "scheduling".
      if (lEnd == 0) {
        lStart = 0;
        lEnd = 1;
        rStart = 0; 
      } else {
        // Move "up" and "left"
        lStart++;
        lEnd++;
        
        rStart -= recurBatchSize;
        
        // Over the top or off the left, reset to layer 0
        if (lEnd > numLayers || rStart < 0) {
           rStart += (lStart + 1) * recurBatchSize;

           lStart = 0;
           lEnd = 1;
        }
        
        // Off the right, step up
        while (rStart >= seqLength && lEnd <= numLayers) {
           lStart++;
           lEnd++;
           
           rStart -= recurBatchSize;
        }
         
        // Over the top or off the left, done!
        if (lEnd > numLayers || rStart < 0) {
           break;
        }
      }

      rEnd = rStart + recurBatchSize;

      rev_lStart = numLayers - lEnd;
      rev_lEnd = numLayers - lStart;
      rev_rStart = seqLength - rStart - 1;
      rev_rEnd = seqLength - rEnd - 1;
      // printf("lStart %d lEnd %d rStart %d rEnd %d\n", rev_lStart, rev_lEnd,
      //    rev_rStart, rev_rEnd);
      if (rEnd > seqLength) rEnd = seqLength;
      for (int layer = rev_lStart; layer < rev_lEnd; layer++) {                 
          
        for (int i = rev_rStart; i > rev_rEnd; i--) {
          // printf("level %d row %d\n", layer, i);
          cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
          //wait for the upper layer
          if (layer < numLayers-1) {
            cudaErrCheck(hipStreamWaitEvent(stream_h[layer], events_h[layer + 1][i], 0));
            cudaErrCheck(hipEventDestroy(events_h[layer + 1][i]));
          }
          //pointwise operations get diff
          cudaErrCheck(hipDeviceSynchronize());
          dim3 blockDim;
          dim3 gridDim;

          blockDim.x = 256;
          gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;               

          elementWise_bp <<< gridDim, blockDim , 0, stream_h[layer] >>> 
              (hiddenSize, miniBatch,
              y_diff + i * numElements + layer * numElements * seqLength, 
              (i >= seqLength - 1) ? NULL : stateGates_diff + 4 * ((i + 1) * numElements + layer * seqLength * numElements), 
              stateGates_diff + 4 * (i * numElements + layer * seqLength  * numElements), 
              stateGates + 4 * (i * numElements + layer * seqLength  * numElements),
              PEEPHOLES ? peeps + 3 * layer * hiddenSize:NULL,
              PEEPHOLES ? peeps_diff + 3 * layer * numElements:NULL,
              c_data + i * numElements + layer * (seqLength + 1) * numElements,
              c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
              c_diff + layer * numElements,
              i == 0);

          printWeight();

          cudaErrCheck(hipGetLastError());
          // transa = (PRE_TRANSPOSE && (seqLength > 1)) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

          //W*diff = dx
          if (layer > 0) {
            float alpha = 1.f;
            float beta = 0.f; 
            cublasErrCheck(hipblasSgemm(handle,
                        HIPBLAS_OP_T, transb,
                        hiddenSize, miniBatch, 4 * hiddenSize,
                        &alpha,
                        &T_f[4 * hiddenSize * inputSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize], 
                        4 * hiddenSize,
                        stateGates_diff + 4 * (i * numElements + layer * seqLength  * numElements),
                        4 * hiddenSize,
                        &beta,
                        y_diff + (layer - 1) * numElements * seqLength + i * numElements, 
                        hiddenSize));
          }

          if(layer != 0) {
            cudaErrCheck(hipEventCreate(&events_h[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_h[layer][i], stream_h[layer])); 
          } 

          if (i > 0) {
            //RT * diff = dy
            float alpha = 1.f;
            float beta = 1.f;
            cublasErrCheck(hipblasSgemm(handle,
                        HIPBLAS_OP_T, transb,
                        hiddenSize, miniBatch, 4 * hiddenSize,
                        &alpha,
                        layer > 0 ? &T_f[(layer - 1) * 8 * hiddenSize * hiddenSize + 8 * hiddenSize * hiddenSize + 4 * hiddenSize * inputSize]:&T_f[4 * hiddenSize * inputSize], 
                        4 * hiddenSize,
                        stateGates_diff + 4 * (i * numElements + layer * seqLength  * numElements),
                        4 * hiddenSize,
                        &beta,
                        y_diff + layer * numElements * seqLength + (i - 1) * numElements, 
                        hiddenSize));
            cudaErrCheck(hipEventCreate(&events_i[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_i[layer][i], stream_h[layer])); 

          } 
          else {
                
            float lr = -learningRate;

            float beta = 1.f;
            //update W
            cublasErrCheck(hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,
                        4 * hiddenSize, layer > 0 ? hiddenSize : inputSize, miniBatch * seqLength,
                        &lr,
                        stateGates_diff + 4 * layer * seqLength * numElements, 
                        4 * hiddenSize,
                        layer > 0 ? i_data + (layer - 1) * seqLength * numElements + seqLength * inputNumElements : i_data,
                        layer > 0 ? hiddenSize : inputSize,
                        &beta,
                        layer > 0 ? &T_f[4 * hiddenSize * inputSize + 4 * hiddenSize * hiddenSize + (layer - 1) * 8 * hiddenSize * hiddenSize]:&T_f[0], 
                        4 * hiddenSize));


            // cudaErrCheck(hipStreamWaitEvent(stream_i[layer], events_i[layer][i+1], 0));
            // cudaErrCheck(hipEventDestroy(events_i[layer][i+1]));

            // cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
            //update R
            cublasErrCheck(hipblasSgemm(handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_T,
                        4 * hiddenSize, hiddenSize, miniBatch * (seqLength - 1),
                        &lr,
                        stateGates_diff + 4 *  (layer * seqLength  * numElements + numElements), 
                        4 * hiddenSize,
                        i_data + layer * seqLength * numElements + seqLength * inputNumElements,
                        hiddenSize,
                        &beta,
                        layer > 0 ? &T_f[(layer - 1) * 8 * hiddenSize * hiddenSize + 8 * hiddenSize * hiddenSize + 4 * hiddenSize * inputSize]:&T_f[4 * hiddenSize * inputSize], 
                        4 * hiddenSize));

            //update bias
            cublasErrCheck(hipblasSgemv(handle,
                        HIPBLAS_OP_N, 
                        4 * hiddenSize, miniBatch * seqLength, 
                        &lr,
                        stateGates_diff + 4 *  (layer * seqLength  * numElements), 
                        4 * hiddenSize,
                        diff_helper,
                        1,
                        &beta,
                        &bias[layer * hiddenSize * 4], 
                        1));

            if (PEEPHOLES) {
              //update peeps
              cublasErrCheck(hipblasSgemv(handle,
                        HIPBLAS_OP_N, 
                        3 * hiddenSize, miniBatch * seqLength, 
                        &lr,
                        peeps_diff + 3 *  (layer  * numElements), 
                        3 * hiddenSize,
                        diff_helper,
                        1,
                        &beta,
                        &peeps[layer * hiddenSize * 3], 
                        1));
            }
          }
        }
      }
    }
    cudaErrCheck(hipEventRecord(stop_bp));
    cudaErrCheck(hipEventSynchronize(stop_bp));
    cudaErrCheck(hipEventElapsedTime(&elapsedTime, start_bp, stop_bp));
    
    cudaErrCheck(hipDeviceSynchronize());
    return elapsedTime;
  }

  void printChecksum() {

    float* testOutputi;
    // float* testOutputh;
    float* testOutputc;
    
    
    int numElements = hiddenSize * miniBatch;
    testOutputi = (float*)malloc(numElements * seqLength * sizeof(float));
    // testOutputh = (float*)malloc(numElements * numLayers * sizeof(float));
    testOutputc = (float*)malloc(numElements * numLayers * sizeof(float));

    cudaErrCheck(hipMemcpy(testOutputi, i_data + seqLength * inputNumElements + (numLayers-1) * seqLength * numElements, seqLength * numElements * sizeof(float), hipMemcpyDeviceToHost));
    for (int layer = 0; layer < numLayers; layer++) {
       // cudaErrCheck(hipMemcpy(testOutputh + layer * numElements, h_data + seqLength * numElements + layer * (seqLength + 1) * numElements, numElements * sizeof(float), hipMemcpyDeviceToHost));
       cudaErrCheck(hipMemcpy(testOutputc + layer * numElements, c_data + seqLength * numElements + layer * (seqLength + 1) * numElements, numElements * sizeof(float), hipMemcpyDeviceToHost));
    }
    double checksumi = 0.;
    // double checksumh = 0.;
    double checksumc = 0.;
    
    for (int m = 0; m < miniBatch; m++) {
       for (int j = 0; j < seqLength; j++) {
          for (int i = 0; i < hiddenSize; i++) {
             checksumi += testOutputi[j * numElements + m * hiddenSize + i];
             printf("i: (%d,%d): %f\n", j, i, testOutputi[j * numElements + m * hiddenSize + i]);
          }
       }
       for (int j = 0; j < numLayers; j++) {
          for (int i = 0; i < hiddenSize; i++) {         
             // checksumh += testOutputh[j * numElements + m * hiddenSize + i];
             checksumc += testOutputc[j * numElements + m * hiddenSize + i];
          }
       }
       
       if (m == 0) printf("i checksum (example %d) %E\n", m, checksumi);
       // if (m == 0) printf("h checksum (example %d) %E\n", m, checksumh);
       if (m == 0) printf("c checksum (example %d) %E\n", m, checksumc);
    }
    
    printf("i checksum %f     ", checksumi);
    printf("c checksum %E     \n", checksumc);
    // printf("h checksum %E\n", checksumh);
    
    free(testOutputi);
    free(testOutputc);
    // free(testOutputh);
    cudaErrCheck(hipDeviceSynchronize());
  }

  void printWeight() {
    int t_size = (4 * inputSize * hiddenSize + 4 * hiddenSize * hiddenSize);
    int bias_size = 4 * hiddenSize * numLayers;
    int stats_size = 4 * numElements * seqLength;

    float* t_output, * bias_output, * states_output, * y_output;
    t_output = (float*)malloc( t_size * sizeof(float));
    bias_output = (float*)malloc(bias_size * sizeof(float));
    states_output = (float*)malloc(8 * sizeof(float));
    y_output = (float*)malloc(2 * sizeof(float));

    cudaErrCheck(hipMemcpy(t_output, T_f, t_size * sizeof(float),hipMemcpyDeviceToHost));  
    cudaErrCheck(hipMemcpy(bias_output,bias,bias_size * sizeof(float),hipMemcpyDeviceToHost));  
    cudaErrCheck(hipMemcpy(states_output,stateGates_diff,stats_size * sizeof(float),hipMemcpyDeviceToHost));  
    cudaErrCheck(hipMemcpy(y_output, y_diff, seqLength*sizeof(float),hipMemcpyDeviceToHost));  

    for (int i = 0; i < t_size;i++) {
      printf("w/r(%d):%f\t",i,t_output[i]);
    }
    printf("\n");
    for (int i = 0; i < bias_size; i++) {
      printf("bias(%d):%f\t",i,bias_output[i]);
    }
    printf("\n");
    for (int i = 0; i < stats_size; i++) {
      printf("%f\t",i,states_output[i]);
    }
    printf("\ny");
    for (int i = 0; i < seqLength; i++) {
      printf("%f\t",i,y_output[i]);
    }
    printf("\n");

  }

  void freeMemory() {
      // cudaErrCheck(hipFree(h_data));
    cudaErrCheck(hipFree(i_data));  
    cudaErrCheck(hipFree(c_data));  

    if (T != T_f) cudaErrCheck(hipFree(T)); 
    cudaErrCheck(hipFree(T_f));

    cudaErrCheck(hipFree(bias));

    cudaErrCheck(hipFree(loss));
    cudaErrCheck(hipFree(label));
    cudaErrCheck(hipFree(mask));
    

    if (PEEPHOLES) {
      cudaErrCheck(hipFree(peeps));
    }
    cudaErrCheck(hipFree(tmp_h));
    cudaErrCheck(hipFree(tmp_i));


    if (TRAINING) {
      // cudaErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
      cudaErrCheck(hipFree(stateGates));
      cudaErrCheck(hipFree(stateGates_diff));
      cudaErrCheck(hipFree(y_diff));
      cudaErrCheck(hipFree(c_diff));
      cudaErrCheck(hipFree(T_diff));
      cudaErrCheck(hipFree(bias_diff));
      if (PEEPHOLES)
        cudaErrCheck(hipFree(peeps_diff));
      cudaErrCheck(hipFree(diff_helper));

    }

    for (int i = 0; i < numLayers; i++) {
      if (stream_i[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_i[i]));
      if (stream_h[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_h[i]));
    }

    free(stream_i);
    free(stream_h);

    for (int i = 0; i < numLayers; i++) {
      free(events_i[i]);
      free(events_h[i]);
    }

    free(events_i);
    free(events_h);
  }
   
};




float LSTMTest(int hiddenSize, int miniBatch, int seqLength, int numLayers, int inputSize, bool checkF) {

  float loss; 
  float elapsedTime;  
  LSTM_scheduler scheduler(hiddenSize,miniBatch,seqLength,numLayers,inputSize);


  scheduler.init();
  printf("Initialize success\n");

  
  scheduler.Forward(&loss);
  printf("Forward loss is %f\n", loss);


  // if (checkF) {
  //   scheduler.printChecksum();
  // }


  
  if(TRAINING) {
    scheduler.clearStates();
    elapsedTime = scheduler.Backward(0.2);
    printf("Backward time is %f\n", elapsedTime);
  }

  scheduler.printWeight();

  // // Timing starts here
  
  // elapsedTime = scheduler.Forward(&loss);
  // printf("Forward time is %f, loss is %f\n", elapsedTime, loss);

  // We're done. Print some checksums
  // if (checkF) {
    // scheduler.printChecksum();
  // }

  scheduler.freeMemory();

  return 0;
}


int main(int argc, char* argv[]) {
  int seqLength;
  int numLayers;
  int hiddenSize;
  int miniBatch; 
  int inputSize;

  if (argc == 6) {
    seqLength = atoi(argv[1]);
    numLayers =  atoi(argv[2]);
    hiddenSize =  atoi(argv[3]);
    miniBatch =  atoi(argv[4]);   
    inputSize = atoi(argv[5]);
  }
  else if (argc == 1) {
    printf("Running with default settings\n");
    inputSize = 1;
    seqLength = 2;
    numLayers = 1;
    hiddenSize = 1;
    miniBatch = 1;
  }
  else {
    printf("Usage: ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch> <inputSize>\n");
    return 1;      
  }

  printf("seqLength %d, numLayers %d, hiddenSize %d, miniBatch %d inputSize %d\n", seqLength, numLayers, hiddenSize, miniBatch, inputSize);  

  int numRuns = 1;

  float totalTime = 0.f;
  for (int run = 0; run < numRuns; run++) {
    totalTime += LSTMTest(hiddenSize, miniBatch, seqLength, numLayers, inputSize, true);
  }

  // printf("Runtime %fms\n", totalTime / numRuns);
  return time < 0;
}

